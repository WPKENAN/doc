#include "hip/hip_runtime.h"
#include "iostream"
#include "stdio.h"
#include "time.h"
#include<Windows.h>
// #include "o"
using namespace std;
int N=10;
#define THREDS_NUMBER 1

__global__ void add(int *a, int *b,int N){
  int tid = threadIdx.x;  //计算机该索引处的数据
  // printf("**************tid: %d\n",tid);
  int cnt=0;
  // printf("N: %d",N*N);
  while(tid<N*N){
  	cnt++;

  	if(tid/N==0||tid/N==N-1||tid%N==0||tid%N==N-1){
  		b[tid]=0;
  	}else{
  		b[tid]=a[tid-N]+a[tid+N]+a[tid-1]+a[tid+1];
  	}
  	tid=tid+THREDS_NUMBER;
  }

}

__global__ void kernal(void){
	// cout << blockIdx.x << endl;
	printf("blockIdx.x: %d Hello cudaKernal!\n",blockIdx.x);
	// printf("");

}


int main(){


	LARGE_INTEGER timeStart;	//开始时间
	LARGE_INTEGER timeEnd;		//结束时间
	LARGE_INTEGER frequency;	//计时器频率

	QueryPerformanceFrequency(&frequency);
	double quadpart = (double)frequency.QuadPart;//计时器频率	

	while(N<=1e7){
		// cout << "Please input the scale of matrix: " ;
		// cin >> N;
		cout << "The scale of matrix: " << N << "^2" << endl;
		// cin >> N;
		clock_t startTime,endTime;

		int *a,*b;
		a=(int*)malloc(N*N*sizeof(int));
		b=(int*)malloc(N*N*sizeof(int));
		// cout << "The run time is:" << ((DWORD)GetTickCount() - start_time) << "ms!" << endl;

		int *dev_a, *dev_b;
		// freopen("out.txt","w",stdout);
		//在GPU上分配内存，注意这里要知道为什么使用void**
		hipMalloc( (void**)&dev_a, N*N*sizeof(int));
		hipMalloc( (void**)&dev_b, N*N*sizeof(int));

		// cout << "The run time is:" << ((DWORD)GetTickCount() - start_time) << "ms!" << endl;
		
		// printf("start\n");
		//创建a值
		for(int i=0;i<N*N;i++){
			a[i]=i;
		}
		// cout << "The run time is:" << ((DWORD)GetTickCount() - start_time) << "ms!" << endl;
		// for(int i=0;i<N;i++){
		// 	for(int j=0;j<N;j++){/
		
		// 		cout << a[i*N+j] << ' ';
		// 	}
		// 	cout << endl;
		// }

		// Sleep(100);
		// //将数组a，b复制到GPU
		hipMemcpy(dev_a, a, N*N*sizeof(int),hipMemcpyHostToDevice);
		hipMemcpy(dev_b, b, N*N*sizeof(int), hipMemcpyHostToDevice);
		// cout << "start" << endl;
		// clock_t start_time = clock();

		QueryPerformanceCounter(&timeStart);
		add<<<1,THREDS_NUMBER>>>(dev_a, dev_b,N);
		QueryPerformanceCounter(&timeEnd);
		// cout << quadpart << endl;
		double elapsed = (timeEnd.QuadPart - timeStart.QuadPart) / quadpart*1000;
		cout << elapsed << "ms" << endl;//单位为秒，精度为微秒(1000000/cpu主频)

		// cout << CLOCKS_PER_SEC << endl;
		// cout << "The run time is:" << (clock() - start_time) << "ms!" << endl;
		// cout << "TimeSpend: " << endTime-startTime << "ms" << endl;
		hipMemcpy(b, dev_b, N*N*sizeof(int), hipMemcpyDeviceToHost);

		// cout << "****************************************" << endl;
		// cout << "The run time is:" << ((DWORD)GetTickCount() - start_time) << "ms!" << endl;
		//显式结果
		// for(int i=0;i<N;i++){
		// 	for(int j=0;j<N;j++){
		// 		cout << b[i*N+j] << ' ';
		// 	}
		// 	cout << endl;
		// }
		//释放GPU上分配的内存
		hipFree(dev_a);
		hipFree(dev_b);

		
		// cout << "The run time is:" << ((DWORD)GetTickCount() - start_time) << "ms!" << endl;
		// cout << endTime << endl;
		// endTime=clock();
		N=N*10;
  	
	}
	// system("pause");
	return 0;
}